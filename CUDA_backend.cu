#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void supCUDA(char* key,char* initaddr,int itersize, int rounds ) 
{
    int id = (threadIdx.x) + (blockIdx.x * 32);
    
    unsigned long long int  hostaddr =(unsigned long long int)initaddr ;

    hostaddr = hostaddr + (id*128);
    char* thread_addr = (char*)hostaddr;


    while (rounds >0)
    {
        //ENCRYPTION BEGINS HERE (128 bytes from thread_addr with 128 bytes of key)  
        for(int i=0 ; i<128 ; i++)
        {
            thread_addr[i] = thread_addr[i]^key[i];
        } 
        //ENCRYPTION ENDS HERE
        thread_addr = thread_addr+itersize;
        rounds--;
    }
    
}

void sequentiel(char* key,char* data,int rounds)
{
    while(rounds>0 || rounds !=0)
    {
        for(int i=0; i<128;i++)
        {
            data[i]= data[i] ^ key[i];
        }
        data = data +128;
        rounds -- ;
    }

}

int main(int argc, char *argv[])
{

    char* argt =  argv[1];
    FILE* tempfile = fopen(argt,"r");
    char line[257];
    char *lines[5]; // Array to store pointers to each line string
    int line_count = 0;

    // Read and store lines
    while (fgets(line, 257, tempfile) != NULL && line_count < 5) 
    {
        // Remove the newline character at the end of the line
        line[strcspn(line, "\n")] = '\0';

        // Skip empty lines
        if (strlen(line) == 0) {
            continue;
        }

        // Allocate memory for the line string and copy the line
        lines[line_count] = (char*)malloc((strlen(line) + 1) * sizeof(char));
        strcpy(lines[line_count], line);
        line_count++;
    }

    fclose(tempfile);


    unsigned long long int temp,residue,cuda_malloc_size,filesize,iter_size,max_free,available_mem,residue_offset,kernel_rounds=0;
    long long int rounds=0;
    char *CudaData, *CudaKey ,*inputfile, *outputfile,*size,*random;//*mode;

    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    inputfile = lines[1];
    outputfile = lines[2];
    size = lines[3];
    // mode = lines[4];

    FILE *reader = fopen(inputfile,"rb");
    FILE* writer = fopen(outputfile,"wb");

    int blocks,threads;
    blocks = deviceProp.multiProcessorCount;
    threads =deviceProp.warpSize;

    filesize  = strtoull(size,&random,10);
    
    hipMemGetInfo(&available_mem,&temp);
    char *key = lines[0] ;

    // printf("inputfile :%s\n",inputfile);
    // printf("outputfile :%s\n",outputfile);
    // printf("key :%s\n",key);
    // printf("size :%llu\n",filesize);
    // printf("mode :%s\n",mode);
    // printf("============================\n");


    hipMalloc((void**)&CudaKey, 129);
    hipMemcpy(CudaKey, key, 128, hipMemcpyHostToDevice);

    iter_size =blocks*threads*128 ;
    residue = available_mem % iter_size;
    max_free = available_mem-residue;

    // printf("\nfree mem:  %llu",available_mem);
    // printf("\nmaxfree:  %llu",max_free);
    // printf("\nitersize:  %llu",iter_size);
    // printf("\nprocessors:  %d",blocks);
    // printf("\nwarp size:  %d",threads);
    // printf("\n=====================\n");

    char *hostptr,*hostptrcpy;

    hostptr = (char*)malloc(filesize);
    fread(hostptr, filesize,1,reader);

    hostptrcpy = hostptr;

    residue = 0;

    if(filesize>max_free)
    {
        cuda_malloc_size=max_free;
        rounds = filesize / max_free;
        residue = filesize % max_free;
        residue_offset = rounds * max_free;
        kernel_rounds = max_free / iter_size;

        hipMalloc((void**)&CudaData, cuda_malloc_size);

        while(rounds>0 || rounds != 0)
        {
            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);

            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();

            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
            hostptrcpy = hostptrcpy + max_free;

            rounds--;


        }
        hipFree(CudaData);

    }
    else
    {
        rounds=0; 

        if(filesize <= iter_size)
        {
            residue = filesize;
            residue_offset = 0;
            kernel_rounds = 0;

        }
        else
        {

            temp =  filesize/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = filesize - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;

            hipMalloc((void**)&CudaData, cuda_malloc_size);
            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);

            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();

            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
            hipFree(CudaData);

        }
        
    }


    if(residue!=0)
    {

        if(residue >= iter_size*10)
        {
            temp =  residue/iter_size;
            cuda_malloc_size = temp * iter_size;
            residue = residue - cuda_malloc_size;
            residue_offset = cuda_malloc_size;
            kernel_rounds = temp ;

            hipMalloc((void**)&CudaData, cuda_malloc_size);
            hipMemcpy(CudaData,hostptrcpy,cuda_malloc_size,hipMemcpyHostToDevice);

            //Launch CUDA kernel here
            supCUDA<<<blocks,threads>>>(CudaKey,CudaData,iter_size,kernel_rounds);
            hipDeviceSynchronize();

            hipMemcpy(hostptrcpy,CudaData,cuda_malloc_size,hipMemcpyDeviceToHost);
            hipFree(CudaData);
            hostptrcpy = hostptrcpy + residue_offset;

        }

        if(residue > 0 )
        {

            if((residue%128)==0)
            {
                rounds = residue / 128;
            }
            else
            {
                temp = residue%128 ;
                residue = residue + (128-temp);
                rounds = residue/128 ;

            }


            hostptrcpy= hostptr + residue_offset;

            // printf("\n=====================\n");
            // printf("residue:%lld\n",residue);
            // printf("Case4 light sequential execution\n");
            // printf("Sequential rounds: %lld\n",rounds);

            //Launch normal kernel here
            sequentiel(key,hostptrcpy,rounds);

        }


    }
    hipFree(CudaKey);

    fwrite(hostptr, filesize,1,writer);

    fclose(writer);
    fclose(reader);

    for (int i = 0; i < line_count; i++) 
    {
        free(lines[i]);
    }
    free(hostptr);

    return 0;
}
