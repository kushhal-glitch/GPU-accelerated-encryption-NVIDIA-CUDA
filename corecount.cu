#include<hip/hip_runtime.h>

int main()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    unsigned long long int blocks, threads;

    blocks = deviceProp.multiProcessorCount;
    threads =deviceProp.warpSize;
    return threads*blocks;
}